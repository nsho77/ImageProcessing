#include "hip/hip_runtime.h"


#include <stdio.h>

// gpu에 할당할 메모리를 저장할 변수를 선언한다.
// 하나는 호스트에서 받아올 용도 다른 하나는 호스트로 넘겨줄 용도
unsigned char* g_tempBuffer[2] = { nullptr };

// gpu에 할당된 메모리를 0으로 세팅하는 함수를 정의한다.
// 영상처리 할 것이므로 width와 height를 인자로 받는다.
int ImageProc_InitializeMemory(int width, int height)
{
	hipError_t cudaStatus;
	cudaStatus = hipMemset(g_tempBuffer[0], 0,
		sizeof(unsigned char)*width*height);
	// 첫번째 메모리 세팅에 실패하면 -1을 반환한다.
	if (cudaStatus != hipSuccess)
		return -1;

	cudaStatus = hipMemset(g_tempBuffer[1], 0,
		sizeof(unsigned char)*width*height);
	// 두번째 메모리 세팅에 실패하면 -2을 반환한다.
	if (cudaStatus != hipSuccess)
		return -2;

	return 1;
}

// gpu에 메모리를 할당하는 함수를 정의한다.
// 해당 기능은 외부에서 쓰이므로 extern "C" 선언을 해준다.
extern "C"
int ImageProc_AllocGPUMemory(int width, int height)
{
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**)&g_tempBuffer[0], 
		sizeof(unsigned char)*width*height);
	// 첫번째 메모리 할당에 실패하면 -1을 반환한다.
	if (cudaStatus != hipSuccess)
		return -1;

	cudaStatus = hipMalloc((void**)&g_tempBuffer[1],
		sizeof(unsigned char)*width*height);
	// 두번째 메모리 할당에 실패하면 -2을 반환한다.
	if (cudaStatus != hipSuccess)
		return -2;

	// 할당한 메모리를 0 으로 세팅한다.
	return ImageProc_InitializeMemory(width, height);
}

extern "C"
// gpu에 할당한 메모리를 반환하는 기능을 정의한다.
int ImageProc_DeAllocGPUMemory(void)
{
	hipError_t cudaStatus;
	cudaStatus = hipFree(g_tempBuffer[0]);
	// 첫번째 메모리 반환에 실패하면 -1을 반환한다.
	if (cudaStatus != hipSuccess)
		return -1;

	cudaStatus = hipFree(g_tempBuffer[1]);
	// 두번째 메모리 반환에 실패하면 -2을 반환한다.
	if (cudaStatus != hipSuccess)
		return -2;

	return 1;
}

extern "C"
int ImageProc_AdaptiveBinarization(unsigned char* image_gray,
	int width, int height, int ksize)
{
	// GPU에 메모리 할당하고 0으로 세팅한다.
	if (ImageProc_AllocGPUMemory(width, height) < 0)
		return -1;

	// GPU 메모리에 호스트 데이터 복사한다.
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(g_tempBuffer[0],image_gray,
		sizeof(unsigned char)*width*height, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		return -2;

	// 커널 함수를 실행한다.


	// 커널 함수 모두 종료를 확인한다.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
		return -4;

	// 결과를 호스트 메모리로 복사한다.
	cudaStatus = hipMemcpy(image_gray, g_tempBuffer[1],
		sizeof(unsigned char)*width*height, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		return -5;

	return 1;
	
}