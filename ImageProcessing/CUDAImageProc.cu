#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

// gpu�� �Ҵ��� �޸𸮸� ������ ������ �����Ѵ�.
// �ϳ��� ȣ��Ʈ���� �޾ƿ� �뵵 �ٸ� �ϳ��� ȣ��Ʈ�� �Ѱ��� �뵵
unsigned char* g_tempBuffer[2] = { nullptr };

// gpu�� �Ҵ�� �޸𸮸� 0���� �����ϴ� �Լ��� �����Ѵ�.
// ����ó�� �� ���̹Ƿ� width�� height�� ���ڷ� �޴´�.
int ImageProc_InitializeMemory(int width, int height)
{
	hipError_t cudaStatus;
	cudaStatus = hipMemset(g_tempBuffer[0], 0,
		sizeof(unsigned char)*width*height);
	// ù��° �޸� ���ÿ� �����ϸ� -1�� ��ȯ�Ѵ�.
	if (cudaStatus != hipSuccess)
		return -1;

	cudaStatus = hipMemset(g_tempBuffer[1], 0,
		sizeof(unsigned char)*width*height);
	// �ι�° �޸� ���ÿ� �����ϸ� -2�� ��ȯ�Ѵ�.
	if (cudaStatus != hipSuccess)
		return -2;

	return 1;
}

// gpu�� �޸𸮸� �Ҵ��ϴ� �Լ��� �����Ѵ�.
// �ش� ����� �ܺο��� ���̹Ƿ� extern "C" ������ ���ش�.
extern "C"
int ImageProc_AllocGPUMemory(int width, int height)
{
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**)&g_tempBuffer[0], 
		sizeof(unsigned char)*width*height);
	// ù��° �޸� �Ҵ翡 �����ϸ� -1�� ��ȯ�Ѵ�.
	if (cudaStatus != hipSuccess)
		return -1;

	cudaStatus = hipMalloc((void**)&g_tempBuffer[1],
		sizeof(unsigned char)*width*height);
	// �ι�° �޸� �Ҵ翡 �����ϸ� -2�� ��ȯ�Ѵ�.
	if (cudaStatus != hipSuccess)
		return -2;

	// �Ҵ��� �޸𸮸� 0 ���� �����Ѵ�.
	return ImageProc_InitializeMemory(width, height);
}

extern "C"
// gpu�� �Ҵ��� �޸𸮸� ��ȯ�ϴ� ����� �����Ѵ�.
int ImageProc_DeAllocGPUMemory(void)
{
	hipError_t cudaStatus;
	cudaStatus = hipFree(g_tempBuffer[0]);
	// ù��° �޸� ��ȯ�� �����ϸ� -1�� ��ȯ�Ѵ�.
	if (cudaStatus != hipSuccess)
		return -1;

	cudaStatus = hipFree(g_tempBuffer[1]);
	// �ι�° �޸� ��ȯ�� �����ϸ� -2�� ��ȯ�Ѵ�.
	if (cudaStatus != hipSuccess)
		return -2;

	return 1;
}

__global__ void Kernel_AdaptiveBinarization(unsigned char* image_gray,
	unsigned char* output_image, int width, int height, int ksize)
{
	if (ksize == 1 || ksize % 2 == 0) return;
	int neighbor = ksize / 2;
	// blockIdx, blockDim, threadIdx �� ��ǥ�� ã�´�.
	// blockDim �� block ������ �̹Ƿ� �Ʒ��� ���� ��ǥ�� ���� �� �ִ�.
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	int j = blockIdx.y*blockDim.y+threadIdx.y;

	float avg = 0.f;
	float cnt = 0.f;

	for (int x = -neighbor; x <= neighbor; x++)
	{
		for (int y = -neighbor; y <= neighbor; y++)
		{
			if (i + x < 0 || i + x >= width || j + y < 0 || j + y >= height)
				continue;
			avg += image_gray[width*(j + y) + (i + x)];
			cnt += 1.f;
		}
	}
	avg = avg / cnt;
	if (image_gray[width*j + i] > avg)
		output_image[width*j + i] = 255;
	else
		output_image[width*j + i] = 0;
}


__global__ void Kernel_BinaryDilation(unsigned char* image_binary,
	unsigned char* output_image, int width, int height, int ksize)
{
	if (ksize == 1 || ksize % 2 == 0) return;
	int neighbor = ksize / 2;
	// blockIdx, blockDim, threadIdx �� ��ǥ�� ã�´�.
	// blockDim �� block ������ �̹Ƿ� �Ʒ��� ���� ��ǥ�� ���� �� �ִ�.
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	// ���� �ȼ��� 255 �̸� ������ �۾��� ���� �ʴ´�.
	if (image_binary[width*j + i] == 255) return;

	// �ֺ� �ȼ��� 255 �̸� ���� �ȼ��� 255�� �ٲ۴�.
	for (int x = -neighbor; x <= neighbor; x++)
	{
		for (int y = -neighbor; y <= neighbor; y++)
		{
			if (i + x < 0 || i + x >= width || j + y < 0 || j + y >= height)
				continue;
			if (image_binary[width*(y + j) + i + x] == 255)
			{
				output_image[width*j + i] = 255;
				return;
			}
			
		}
	}

}

__global__ void Kernel_BinaryErosion(unsigned char* image_binary,
	unsigned char* output_image, int width, int height, int ksize)
{
	if (ksize == 1 || ksize % 2 == 0) return;
	int neighbor = ksize / 2;
	// blockIdx, blockDim, threadIdx �� ��ǥ�� ã�´�.
	// blockDim �� block ������ �̹Ƿ� �Ʒ��� ���� ��ǥ�� ���� �� �ִ�.
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	// ���� �ȼ��� 0 �̸� ������ �۾��� ���� �ʴ´�.
	if (image_binary[width*j + i] == 0) return;

	// �ֺ� �ȼ��� 0 �̸� ���� �ȼ��� 0�� �ٲ۴�.
	for (int x = -neighbor; x <= neighbor; x++)
	{
		for (int y = -neighbor; y <= neighbor; y++)
		{
			if (i + x < 0 || i + x >= width || j + y < 0 || j + y >= height)
				continue;
			if (image_binary[width*(y + j) + i + x] == 0)
			{
				output_image[width*j + i] = 0;
				return;
			}

		}
	}

}

extern "C"
int ImageProc_AdaptiveBinarization(unsigned char* image_gray,
	int width, int height, int ksize)
{
	// GPU�� �޸� �Ҵ��ϰ� 0���� �����Ѵ�.
	/*if (ImageProc_AllocGPUMemory(width, height) < 0)
		return -1;*/

	// GPU �޸𸮿� ȣ��Ʈ ������ �����Ѵ�.
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(g_tempBuffer[0],image_gray,
		sizeof(unsigned char)*width*height, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		return -2;

	// Ŀ�� �Լ��� �����Ѵ�.
	// GridDim.x, GridDim.y, BlockDim.x, BlockDim.y �� �����Ѵ�.
	dim3 Db = dim3(8,8);
	dim3 Dg = dim3((width + Db.x - 1) / Db.x, (height + Db.y - 1) / Db.y);
	Kernel_AdaptiveBinarization<<< Dg, Db >>> (g_tempBuffer[0],
		g_tempBuffer[1],width,height,ksize);

	// Ŀ�� �Լ� ������ ����� �Ǿ����� Ȯ���Ѵ�.
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		return -3;


	// Ŀ�� �Լ� ��� ���Ḧ Ȯ���Ѵ�.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaStatus : %d\n", cudaStatus);
		return -4;
	}
		

	// ����� ȣ��Ʈ �޸𸮷� �����Ѵ�.
	cudaStatus = hipMemcpy(image_gray, g_tempBuffer[1],
		sizeof(unsigned char)*width*height, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		return -5;

	return 1;
	
}

extern "C"
int ImageProc_BinaryDilation(unsigned char* image_binary,
	int width, int height, int ksize)
{
	// GPU �޸𸮿� ȣ��Ʈ ������ �����Ѵ�.
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(g_tempBuffer[0], image_binary,
		sizeof(unsigned char)*width*height, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		return -2;

	cudaStatus = hipMemcpy(g_tempBuffer[1], image_binary,
		sizeof(unsigned char)*width*height, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		return -2;

	// Ŀ�� �Լ��� �����Ѵ�.
	// GridDim.x, GridDim.y, BlockDim.x, BlockDim.y �� �����Ѵ�.
	dim3 Db = dim3(8, 8);
	dim3 Dg = dim3((width + Db.x - 1) / Db.x, (height + Db.y - 1) / Db.y);
	Kernel_BinaryDilation << < Dg, Db >> > (g_tempBuffer[0],
		g_tempBuffer[1], width, height, ksize);

	// Ŀ�� �Լ� ������ ����� �Ǿ����� Ȯ���Ѵ�.
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		return -3;


	// Ŀ�� �Լ� ��� ���Ḧ Ȯ���Ѵ�.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaStatus : %d\n", cudaStatus);
		return -4;
	}


	// ����� ȣ��Ʈ �޸𸮷� �����Ѵ�.
	cudaStatus = hipMemcpy(image_binary, g_tempBuffer[1],
		sizeof(unsigned char)*width*height, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		return -5;

	return 1;

}


extern "C"
int ImageProc_BinaryErosion(unsigned char* image_binary,
	int width, int height, int ksize)
{
	// GPU �޸𸮿� ȣ��Ʈ ������ �����Ѵ�.
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(g_tempBuffer[0], image_binary,
		sizeof(unsigned char)*width*height, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		return -2;

	cudaStatus = hipMemcpy(g_tempBuffer[1], image_binary,
		sizeof(unsigned char)*width*height, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		return -2;

	// Ŀ�� �Լ��� �����Ѵ�.
	// GridDim.x, GridDim.y, BlockDim.x, BlockDim.y �� �����Ѵ�.
	dim3 Db = dim3(8, 8);
	dim3 Dg = dim3((width + Db.x - 1) / Db.x, (height + Db.y - 1) / Db.y);
	Kernel_BinaryErosion << < Dg, Db >> > (g_tempBuffer[0],
		g_tempBuffer[1], width, height, ksize);

	// Ŀ�� �Լ� ������ ����� �Ǿ����� Ȯ���Ѵ�.
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		return -3;


	// Ŀ�� �Լ� ��� ���Ḧ Ȯ���Ѵ�.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		printf("cudaStatus : %d\n", cudaStatus);
		return -4;
	}


	// ����� ȣ��Ʈ �޸𸮷� �����Ѵ�.
	cudaStatus = hipMemcpy(image_binary, g_tempBuffer[1],
		sizeof(unsigned char)*width*height, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		return -5;

	return 1;

}